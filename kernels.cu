#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#include "common.h"
#include "kernels.cuh"

__constant__ float d_laplace3[3 * 3] = { 0.00f,  1.00f,  0.00f,
										 1.00f, -4.00f,  1.00f,
										 0.00f,  1.00f,  0.00f };

__device__ float2 distRegPre(float4 gradUnNormalized)
{
	float mag = sqrt(gradUnNormalized.x * gradUnNormalized.x + gradUnNormalized.y * gradUnNormalized.y);
	float a = (mag >= 0.0f) && (mag <= 1.0f);
	float b = (mag > 1.0f);
	float ps = a * sin(2.0f * HIP_PI_F * mag) / (2.0f * HIP_PI_F) + b * (mag - 1.0f);
	float dps = ((ps != 0.0f) * ps + (ps == 0.0f)) / ((mag != 0.0f) * mag + (mag == 0.0f)) - 1.0f;

	return make_float2(dps * gradUnNormalized.x, dps * gradUnNormalized.y);
}

__global__ void edgeIndicatorKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
	// Calculate surface coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float plusX, minX, plusY, minY;

	surf2Dread(&plusX, input, (x + 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&minX, input, (x - 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&plusY, input, x * sizeof(float), y + 1, hipBoundaryModeClamp);
	surf2Dread(&minY, input, x * sizeof(float), y - 1, hipBoundaryModeClamp);

	float gradX = (plusX - minX);
	float gradY = (plusY - minY);

	surf2Dwrite(1.0f / (1.0f + 0.25f * (gradX * gradX + gradY * gradY)),
		output,
		x * sizeof(float),
		y,
		hipBoundaryModeClamp);
}

__global__ void laplaceKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
	// Calculate surface coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float sum = 0.0f;
	float sample;

#pragma unroll
	for (int i = -1; i <= 1; i++)
	{
#pragma unroll
		for (int j = -1; j <= 1; j++)
		{
			surf2Dread(&sample, input, (x + i) * sizeof(float), y + j, hipBoundaryModeClamp);
			sum += sample * d_laplace3[3 * (i + 1) + (j + 1)];
		}
	}

	surf2Dwrite(sum,
		output, x * sizeof(float),
		y,
		hipBoundaryModeClamp);
}

__global__ void gradKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
	// Calculate surface coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float plusX, minX, plusY, minY;

	surf2Dread(&plusX, input, (x + 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&minX, input, (x - 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&plusY, input, x * sizeof(float), y + 1, hipBoundaryModeClamp);
	surf2Dread(&minY, input, x * sizeof(float), y - 1, hipBoundaryModeClamp);

	surf2Dwrite<float2>(make_float2((plusX - minX) * 0.5f, (plusY - minY) * 0.5f),
		output,
		x * sizeof(float2),
		y,
		hipBoundaryModeClamp);
}

__global__ void gradNormKernel(hipSurfaceObject_t input, hipSurfaceObject_t output)
{
	// Calculate surface coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float plusX, minX, plusY, minY;

	surf2Dread(&plusX, input, (x + 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&minX, input, (x - 1) * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&plusY, input, x * sizeof(float), y + 1, hipBoundaryModeClamp);
	surf2Dread(&minY, input, x * sizeof(float), y - 1, hipBoundaryModeClamp);

	float gradX = (plusX - minX) * 0.5f;
	float gradY = (plusY - minY) * 0.5f;

	float gradMag = sqrt(gradX * gradX + gradY * gradY) + 1e-10f; //prevent div by zero

	surf2Dwrite<float4>(make_float4(gradX, gradY, gradX / gradMag, gradY / gradMag),
		output,
		x * sizeof(float4),
		y,
		hipBoundaryModeClamp);
}

__global__ void applyKernel7x7(hipSurfaceObject_t input, float* d_kernel, hipSurfaceObject_t output)
{
	// Calculate surface coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float sum = 0.0f;
	float sample;

#pragma unroll
	for (int i = -3; i <= 3; i++)
	{
#pragma unroll
		for (int j = -3; j <= 3; j++)
		{
			surf2Dread(&sample, input, (x + i) * sizeof(sample), y + j, hipBoundaryModeClamp);
			sum += sample * d_kernel[7 * (i + 3) + (j + 3)];
		}
	}

	surf2Dwrite(sum,
		output, x * sizeof(float),
		y,
		hipBoundaryModeClamp);
}

__global__ void distRegKernel(hipSurfaceObject_t phiGradSurf, hipSurfaceObject_t distRegSurf)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 phiGrad;
	surf2Dread(&phiGrad, phiGradSurf, x * sizeof(float4), y, hipBoundaryModeClamp);

	float phiGradMag = sqrt(phiGrad.x * phiGrad.x + phiGrad.y * phiGrad.y);

	float a = (phiGradMag >= 0.0f) && (phiGradMag <= 1.0f);
	float b = (phiGradMag > 1.0f);
	float ps = a * sin(2.0f * HIP_PI_F * phiGradMag) / (2.0f * HIP_PI_F) + b * (phiGradMag - 1.0f);
	float dps = ((ps != 0.0f) * ps + (ps == 0.0f)) / ((phiGradMag != 0.0f) * phiGradMag + (phiGradMag == 0.0f)) - 1.0f;

	float2 result = make_float2(dps * phiGrad.x, dps * phiGrad.y);

	surf2Dwrite(result,
		distRegSurf,
		x * sizeof(float2),
		y,
		hipBoundaryModeClamp);
}

__global__ void levelSetKernel(float mu, float lambda, float alpha, float epsilon, float timestep,
	hipSurfaceObject_t phiSurf, hipSurfaceObject_t edgeSurf, hipSurfaceObject_t edgeGradSurf,
	hipSurfaceObject_t gradPhiSurf, hipSurfaceObject_t laplaceSurf, hipSurfaceObject_t nextPhiSurf)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	//Read everything we need
	float phi;
	float4 gradPhi;
	surf2Dread(&phi, phiSurf, x * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&gradPhi, gradPhiSurf, x * sizeof(float4), y, hipBoundaryModeClamp);

	float edge;
	float2 edgeGrad;
	surf2Dread(&edge, edgeSurf, x * sizeof(float), y, hipBoundaryModeClamp);
	surf2Dread(&edgeGrad, edgeGradSurf, x * sizeof(float2), y, hipBoundaryModeClamp);

	//Curvature
	float4 plusX, minX, plusY, minY;
	surf2Dread(&plusX, gradPhiSurf, (x + 1) * sizeof(float4), y, hipBoundaryModeClamp);
	surf2Dread(&minX, gradPhiSurf, (x - 1) * sizeof(float4), y, hipBoundaryModeClamp);
	surf2Dread(&plusY, gradPhiSurf, x * sizeof(float4), y + 1, hipBoundaryModeClamp);
	surf2Dread(&minY, gradPhiSurf, x * sizeof(float4), y - 1, hipBoundaryModeClamp);
	float curvature = 0.5f * (plusX.z - minX.z + plusY.w - minY.w);

	//Distance regularization term
	float distRegTerm;
	surf2Dread(&distRegTerm, laplaceSurf, x * sizeof(float), y, hipBoundaryModeClamp);
	distRegTerm += 0.5f * (distRegPre(plusX).x - distRegPre(minX).x + distRegPre(plusY).y - distRegPre(minY).y);

	// Calculate Dirac delta
	float f = (0.5f / epsilon) * (1.0f + cos(HIP_PI_F * phi / epsilon));
	float b = (phi <= epsilon) && (phi >= -epsilon);
	float diracDelta = f * b;

	// Calculate LS increment
	float increment = timestep *
		(mu * distRegTerm +
			lambda * diracDelta * (edgeGrad.x * gradPhi.z + edgeGrad.y * gradPhi.w + edge * curvature) +
			alpha * diracDelta * edge);

	// Write results
	surf2Dwrite(phi + increment,
		nextPhiSurf,
		x * sizeof(float),
		y,
		hipBoundaryModeClamp);
}