#include "hip/hip_runtime.h"
#include <chrono>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "common.h"
#include "utils.h"
#include "kernels.cuh"

using namespace std;

__host__ void edgeIndicator(LevelSetData& lsd)
{
	dim3 threadsPerBlock(32, 32, 1);
	dim3 blocksPerGrid(uint(ceil(lsd.width / 32.0f)), uint(ceil(lsd.height / 32.0f)), 1);

	hipSurfaceObject_t d_blurredImage = utils::createSurface(nullptr, lsd.width, lsd.height, hipChannelFormatKindFloat, 32);
	
	applyKernel7x7<<<blocksPerGrid, threadsPerBlock>>>(lsd.d_inputImage, lsd.d_gaussianKernel, d_blurredImage);
	edgeIndicatorKernel<<<blocksPerGrid, threadsPerBlock>>>(d_blurredImage, lsd.d_edge);

	// Also get the gradient of the edge indicator result, as that is also used in the DRLSE loop
	gradKernel<<<blocksPerGrid, threadsPerBlock>>>(lsd.d_edge, lsd.d_edgeGrad);

	utils::freeSurface(d_blurredImage);
}

// h_inImage should be a width * height array, ideally in the [0,1] range
// h_inout_phi should be a binary image describing the initial zero-level set, 
// with 1.0f for pixels inside the contour and -1.0f for pixels outside
void runCUDA(float* h_inImage, float* h_inout_phi, uint width, uint height)
{
	dim3 threadsPerBlock(32, 32, 1);
	dim3 blocksPerGrid(uint(ceil(width / 32.0f)), uint(ceil(height / 32.0f)), 1);	

	LevelSetData lsd;
	lsd.mu           = 0.2f;
	lsd.lambda       = 0.1f;
	lsd.alpha        = 5.0f;
	lsd.sigma        = 1.0f;
	lsd.timestep     = 1.0f;
	lsd.c0           = 10.0f;
	lsd.epsilon      = 1.5f;
	lsd.maxIterCount = 3000;
	lsd.width        = width;
	lsd.height       = height;

	for (uint i = 0; i < width * height; i++)
		h_inout_phi[i] *= lsd.c0;

	lsd.d_inputImage = utils::createSurface(h_inImage, width, height, hipChannelFormatKindFloat, 32);	
	lsd.d_edge       = utils::createSurface(nullptr, width, height, hipChannelFormatKindFloat, 32);
	lsd.d_edgeGrad   = utils::createSurface(nullptr, width, height, hipChannelFormatKindFloat, 32, 32);
	lsd.d_phi	     = utils::createSurface(h_inout_phi, width, height, hipChannelFormatKindFloat, 32);
	lsd.d_gradPhi	 = utils::createSurface(nullptr, width, height, hipChannelFormatKindFloat, 32, 32, 32, 32);
	lsd.d_nextPhi	 = utils::createSurface(nullptr, width, height, hipChannelFormatKindFloat, 32);
	lsd.d_laplace	 = utils::createSurface(nullptr, width, height, hipChannelFormatKindFloat, 32);

	if (lsd.mu * lsd.timestep >= 0.25f)
		printf("Warning: parameters do not meet Courant-Friedrichs-Lewy condition for numerical stability: mu * timestep < 0.25f\n");

	eee(hipMalloc((void **)&lsd.d_gaussianKernel, 7*7*sizeof(float)));	
	utils::buildGaussianKernel(lsd.d_gaussianKernel, lsd.sigma);

	auto start = std::chrono::high_resolution_clock::now();
	{
		edgeIndicator(lsd);

		for (int i = 0; i < lsd.maxIterCount; i++)
		{
			gradNormKernel<<<blocksPerGrid, threadsPerBlock>>>(lsd.d_phi, lsd.d_gradPhi);

			laplaceKernel<<<blocksPerGrid, threadsPerBlock>>>(lsd.d_phi, lsd.d_laplace);

			levelSetKernel << <blocksPerGrid, threadsPerBlock >> > (
				lsd.mu, lsd.lambda, lsd.alpha, lsd.epsilon, lsd.timestep,
				lsd.d_phi, lsd.d_edge, lsd.d_edgeGrad, lsd.d_gradPhi, lsd.d_laplace, lsd.d_nextPhi);

			//Switch references (these are just long longs)
			auto temp = lsd.d_phi;
			lsd.d_phi = lsd.d_nextPhi;
			lsd.d_nextPhi = temp;
		}

		eee(hipDeviceSynchronize());
	}
	auto duration = std::chrono::high_resolution_clock::now() - start;
	long long ms = std::chrono::duration_cast<std::chrono::microseconds>(duration).count();
	printf("runCUDA executed in %lld microseconds\n", ms);    
    
	eee(hipGetLastError());

	// Move the final phi to the inout host array
	hipResourceDesc phiDesc;
	cudaGetSurfaceObjectResourceDesc(&phiDesc, lsd.d_phi);
	eee(hipMemcpyFromArray(h_inout_phi, phiDesc.res.array.array, 0, 0, width * height * sizeof(float), hipMemcpyDeviceToHost)); 
	
	utils::releaseLevelSetData(lsd);

	eee(hipProfilerStop());
	eee(hipDeviceReset());
}

int main(int argc, char **argv)
{
	printf("Starting\n");
	
	uint width = 256;
	uint height = 256;

	vector<float> inputData(width * height);
	vector<float> outputData(width * height);
	for (uint x = 0; x < width; x++)
	{
		for (uint y = 0; y < height; y++)
		{
			if (x > 30 && x < 70 && y > 30 && y < 70)
			{
				inputData[y * width + x] = 1000.0f;
			}
			else if (pow(x - 200.0f, 2.0f) + pow(y - 200.0f, 2.0f) < 250)
			{
				inputData[y * width + x] = 1000.0f;
			}
			else 
			{
				inputData[y * width + x] = 0.0f;
			}

			if (x > 110 && x < 160 && y > 110 && y < 160)
			{
				outputData[y * width + x] = 1.0f;
			}
			else
			{
				outputData[y * width + x] = -1.0f;
			}
		}
	}

	runCUDA(inputData.data(), outputData.data(), width, height);

	// Select the zero level set from the output data
	for (uint x = 0; x < width; x++)
	{
		for (uint y = 0; y < height; y++)
		{
			float val = outputData[y * width + x];

			outputData[y * width + x] = (val > -0.5f && val < 0.5f)? 1.0f : 0.0f;			
		}
	}

	ofstream fout("input.dat", ios::out | ios::binary);
	fout.write((char*)inputData.data(), inputData.size() * sizeof(inputData[0]));
	fout.close();

	fout = ofstream("output.dat", ios::out | ios::binary);
	fout.write((char*)outputData.data(), outputData.size() * sizeof(outputData[0]));
	fout.close();
}